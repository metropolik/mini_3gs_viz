
#include <hip/hip_runtime.h>
extern "C" __global__
void transform_points(const float* matrix, const float* input_positions, 
                     float* output_positions, int num_points) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= num_points) return;
    
    // Input has 4 components (x, y, z, w)
    int input_offset = idx * 4;
    
    // Load input position
    float x = input_positions[input_offset + 0];
    float y = input_positions[input_offset + 1];
    float z = input_positions[input_offset + 2];
    float w = input_positions[input_offset + 3];
    
    // Matrix multiplication: transformed = matrix * input_position
    // Matrix is 4x4, stored in row-major order
    float tx = matrix[0]  * x + matrix[1]  * y + matrix[2]  * z + matrix[3]  * w;
    float ty = matrix[4]  * x + matrix[5]  * y + matrix[6]  * z + matrix[7]  * w;
    float tz = matrix[8]  * x + matrix[9]  * y + matrix[10] * z + matrix[11] * w;
    float tw = matrix[12] * x + matrix[13] * y + matrix[14] * z + matrix[15] * w;
    
    // Determine output format based on output buffer size
    // If output has 4 components per point, keep homogeneous (no perspective division)
    // If output has 3 components per point, perform perspective division
    
    // Check if we should perform perspective division by examining if output stride is 3 or 4
    // We can infer this by checking the memory layout pattern
    // For now, we'll use a simple heuristic: if the kernel is called with 4-component output,
    // we assume it's the first pass (MV transformation), otherwise it's the second pass (P transformation)
    
    // We need to determine output format - let's assume homogeneous output for now
    // and create a separate kernel for perspective division
    int output_offset = idx * 4;  // Assume 4-component output for now
    
    output_positions[output_offset + 0] = tx;
    output_positions[output_offset + 1] = ty;
    output_positions[output_offset + 2] = tz;
    output_positions[output_offset + 3] = tw;
}

extern "C" __global__
void transform_points_with_perspective(const float* matrix, const float* input_positions, 
                                      float* output_positions, int num_points) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= num_points) return;
    
    // Input has 4 components (x, y, z, w)
    int input_offset = idx * 4;
    int output_offset = idx * 3;  // Output only x, y, z (3 components)
    
    // Load input position
    float x = input_positions[input_offset + 0];
    float y = input_positions[input_offset + 1];
    float z = input_positions[input_offset + 2];
    float w = input_positions[input_offset + 3];
    
    // Matrix multiplication: transformed = matrix * input_position
    // Matrix is 4x4, stored in row-major order
    float tx = matrix[0]  * x + matrix[1]  * y + matrix[2]  * z + matrix[3]  * w;
    float ty = matrix[4]  * x + matrix[5]  * y + matrix[6]  * z + matrix[7]  * w;
    float tz = matrix[8]  * x + matrix[9]  * y + matrix[10] * z + matrix[11] * w;
    float tw = matrix[12] * x + matrix[13] * y + matrix[14] * z + matrix[15] * w;
    
    // Perform perspective division
    float valid_w = (fabsf(tw) < 1e-8f) ? 1e-8f : tw;
    
    // Store transformed positions after perspective division (only x, y, z)
    output_positions[output_offset + 0] = tx / valid_w;
    output_positions[output_offset + 1] = ty / valid_w;
    output_positions[output_offset + 2] = tz / valid_w;
}

// Device function to project a point from view space to NDC
__device__ void project_to_ndc(float vx, float vy, float vz, float vw,
                               const float* proj_matrix,
                               float& ndc_x, float& ndc_y, float& ndc_z) {
    // Apply projection matrix
    float clip_x = proj_matrix[0] * vx + proj_matrix[1] * vy + proj_matrix[2] * vz + proj_matrix[3] * vw;
    float clip_y = proj_matrix[4] * vx + proj_matrix[5] * vy + proj_matrix[6] * vz + proj_matrix[7] * vw;
    float clip_z = proj_matrix[8] * vx + proj_matrix[9] * vy + proj_matrix[10] * vz + proj_matrix[11] * vw;
    float clip_w = proj_matrix[12] * vx + proj_matrix[13] * vy + proj_matrix[14] * vz + proj_matrix[15] * vw;
    
    // Perform perspective division
    float inv_w = 1.0f / (fabsf(clip_w) < 1e-8f ? 1e-8f : clip_w);
    ndc_x = clip_x * inv_w;
    ndc_y = clip_y * inv_w;
    ndc_z = clip_z * inv_w;
}

// Compute 2D covariance matrices and quad parameters for Gaussian splatting
extern "C" __global__
void compute_2d_covariance(const float* view_space_positions,    // View space positions (4 components each)
                          const float* scales,                  // Scale vectors (3 components each)
                          const float* rotations,               // Rotation quaternions (4 components each)
                          const float* mv_matrix,               // Model-view matrix (4x4)
                          const float* proj_matrix,             // Projection matrix (4x4)
                          float* cov2d_data,                    // Output: 2D covariance matrices (3 components: cov[0,0], cov[0,1], cov[1,1])
                          float* quad_params,                   // Output: Quad parameters (5 components: center_x, center_y, radius_x, radius_y, ndc_z)
                          int* visibility_mask,                 // Output: Visibility mask (1 if visible, 0 if culled)
                          float viewport_width,                 // Viewport width in pixels
                          float viewport_height,                // Viewport height in pixels
                          int num_points) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= num_points) return;
    
    // Load Gaussian parameters
    int pos_offset = idx * 4;
    int scale_offset = idx * 3;
    int rot_offset = idx * 4;
    int cov_offset = idx * 3;
    int quad_offset = idx * 5;
    
    // Load view space position
    float vx = view_space_positions[pos_offset + 0];
    float vy = view_space_positions[pos_offset + 1];
    float vz = view_space_positions[pos_offset + 2];
    float vw = view_space_positions[pos_offset + 3];
    
    // Don't cull for performance - render all quads
    // Even if behind camera, still compute parameters
    visibility_mask[idx] = 1;  // Mark all as visible
    
    // Load scale and rotation
    float sx = scales[scale_offset + 0];
    float sy = scales[scale_offset + 1];
    float sz = scales[scale_offset + 2];
    
    float qw = rotations[rot_offset + 0];
    float qx = rotations[rot_offset + 1];
    float qy = rotations[rot_offset + 2];
    float qz = rotations[rot_offset + 3];
    
    // Build 3D covariance matrix: Σ_3D = R * S * S^T * R^T
    // First build rotation matrix R from quaternion
    float r11 = 1.0f - 2.0f * (qy*qy + qz*qz);
    float r12 = 2.0f * (qx*qy - qw*qz);
    float r13 = 2.0f * (qx*qz + qw*qy);
    float r21 = 2.0f * (qx*qy + qw*qz);
    float r22 = 1.0f - 2.0f * (qx*qx + qz*qz);
    float r23 = 2.0f * (qy*qz - qw*qx);
    float r31 = 2.0f * (qx*qz - qw*qy);
    float r32 = 2.0f * (qy*qz + qw*qx);
    float r33 = 1.0f - 2.0f * (qx*qx + qy*qy);
    
    // Compute R * S (rotation times scaling)
    float rs11 = r11 * sx, rs12 = r12 * sy, rs13 = r13 * sz;
    float rs21 = r21 * sx, rs22 = r22 * sy, rs23 = r23 * sz;
    float rs31 = r31 * sx, rs32 = r32 * sy, rs33 = r33 * sz;
    
    // Compute 3D covariance: (R*S) * (R*S)^T
    float cov3d_00 = rs11*rs11 + rs12*rs12 + rs13*rs13;
    float cov3d_01 = rs11*rs21 + rs12*rs22 + rs13*rs23;
    float cov3d_02 = rs11*rs31 + rs12*rs32 + rs13*rs33;
    float cov3d_11 = rs21*rs21 + rs22*rs22 + rs23*rs23;
    float cov3d_12 = rs21*rs31 + rs22*rs32 + rs23*rs33;
    float cov3d_22 = rs31*rs31 + rs32*rs32 + rs33*rs33;
    
    // Apply viewing transformation W to the 3D covariance: Σ' = W * Σ_3D * W^T
    // Extract the 3x3 rotation part of the model-view matrix (upper-left 3x3)
    float w00 = mv_matrix[0], w01 = mv_matrix[1], w02 = mv_matrix[2];
    float w10 = mv_matrix[4], w11 = mv_matrix[5], w12 = mv_matrix[6];
    float w20 = mv_matrix[8], w21 = mv_matrix[9], w22 = mv_matrix[10];
    
    // Compute W * Σ_3D
    float ws00 = w00*cov3d_00 + w01*cov3d_01 + w02*cov3d_02;
    float ws01 = w00*cov3d_01 + w01*cov3d_11 + w02*cov3d_12;
    float ws02 = w00*cov3d_02 + w01*cov3d_12 + w02*cov3d_22;
    float ws10 = w10*cov3d_00 + w11*cov3d_01 + w12*cov3d_02;
    float ws11 = w10*cov3d_01 + w11*cov3d_11 + w12*cov3d_12;
    float ws12 = w10*cov3d_02 + w11*cov3d_12 + w12*cov3d_22;
    float ws20 = w20*cov3d_00 + w21*cov3d_01 + w22*cov3d_02;
    float ws21 = w20*cov3d_01 + w21*cov3d_11 + w22*cov3d_12;
    float ws22 = w20*cov3d_02 + w21*cov3d_12 + w22*cov3d_22;
    
    // Compute (W * Σ_3D) * W^T to get view space covariance
    float cov_view_00 = ws00*w00 + ws01*w01 + ws02*w02;
    float cov_view_01 = ws00*w10 + ws01*w11 + ws02*w12;
    float cov_view_02 = ws00*w20 + ws01*w21 + ws02*w22;
    float cov_view_11 = ws10*w10 + ws11*w11 + ws12*w12;
    float cov_view_12 = ws10*w20 + ws11*w21 + ws12*w22;
    float cov_view_22 = ws20*w20 + ws21*w21 + ws22*w22;
    
    // Project to screen space using Jacobian of perspective projection
    // The projection matrix transforms (x,y,z) to (fx*x/z, fy*y/z) where fx, fy are from proj matrix
    // Extract focal length scaling from projection matrix
    float fx = proj_matrix[0];   // P[0,0] = focal_x / aspect
    float fy = proj_matrix[5];   // P[1,1] = focal_y
    
    // For point (x,y,z) in view space, clip coords are (fx*x, fy*y, ...) before division by w
    // Jacobian J = [[fx/z, 0, -fx*x/z^2], [0, fy/z, -fy*y/z^2]]
    // Use absolute value to ensure consistent coordinate system handling
    float inv_z = 1.0f / fabsf(vz);  // Use absolute value for robustness
    float inv_z2 = inv_z * inv_z;
    
    // Jacobian matrix elements with projection matrix scaling
    // J = [[fx/z, 0, -fx*x/z²], [0, fy/z, -fy*y/z²]]
    float j00 = fx * inv_z;
    float j02 = -fx * vx * inv_z2;  // Added missing negative sign
    float j11 = fy * inv_z;
    float j12 = -fy * vy * inv_z2;  // Added missing negative sign
    
    // Compute 2D covariance: Σ_2D = J * Σ_view * J^T
    // J = [[j00, 0, j02], [0, j11, j12]]
    // Σ_view = [[cov_view_00, cov_view_01, cov_view_02],
    //           [cov_view_01, cov_view_11, cov_view_12],
    //           [cov_view_02, cov_view_12, cov_view_22]]
    float cov2d_00 = j00*j00*cov_view_00 + j02*j02*cov_view_22 + 2.0f*j00*j02*cov_view_02;
    float cov2d_01 = j00*j11*cov_view_01 + j02*j12*cov_view_22 + j00*j12*cov_view_02 + j02*j11*cov_view_01;
    float cov2d_11 = j11*j11*cov_view_11 + j12*j12*cov_view_22 + 2.0f*j11*j12*cov_view_12;
    
    
    // Add a much smaller regularization term to ensure positive definiteness
    // The original 1e-4f was too large and dominating the actual values
    cov2d_00 += 1e-8f;
    cov2d_11 += 1e-8f;
    
    // Store 2D covariance (symmetric, so store upper triangle)
    cov2d_data[cov_offset + 0] = cov2d_00;
    cov2d_data[cov_offset + 1] = cov2d_01;
    cov2d_data[cov_offset + 2] = cov2d_11;
    
    // Compute eigenvalues for quad sizing (3σ bounds)
    // For 2x2 symmetric matrix [[a,b],[b,c]], eigenvalues are:
    // λ = (a+c ± sqrt((a-c)² + 4b²)) / 2
    double trace = (double)cov2d_00 + (double)cov2d_11;
    double det = (double)cov2d_00 * (double)cov2d_11 - (double)cov2d_01 * (double)cov2d_01;
    double discriminant = trace * trace - 4.0 * det;
    
    if (discriminant < 0.0) {
        // Degenerate case, but still render with minimal size
        discriminant = 0.0;
    }
    
    double sqrt_disc = sqrt(discriminant);
    float lambda1 = (float)(0.5 * (trace + sqrt_disc));
    float lambda2 = (float)(0.5 * (trace - sqrt_disc));
    
    // Compute radii (3σ = 3 * sqrt(eigenvalue))
    // Note: eigenvalues are now in NDC space due to projection matrix scaling in Jacobian
    float radius_x = 3.0f * sqrtf(fmaxf(lambda1, 1e-6f));
    float radius_y = 3.0f * sqrtf(fmaxf(lambda2, 1e-6f));
    
    
    // Don't cull small quads - render everything
    // Ensure minimum size for degenerate cases
    radius_x = fmaxf(radius_x, 1e-6f);
    radius_y = fmaxf(radius_y, 1e-6f);
    
    // Project center to normalized device coordinates (NDC) using full projection
    float ndc_x, ndc_y, ndc_z;
    project_to_ndc(vx, vy, vz, vw, proj_matrix, ndc_x, ndc_y, ndc_z);
    
    // Handle invalid values by clamping
    if (!isfinite(ndc_x)) ndc_x = 0.0f;
    if (!isfinite(ndc_y)) ndc_y = 0.0f;
    if (!isfinite(ndc_z)) ndc_z = 0.0f;
    
    // The radii are already in the correct space due to projection matrix scaling in Jacobian
    // No additional conversion needed
    float radius_x_ndc = radius_x;
    float radius_y_ndc = radius_y;
    
    
    // No artificial capping needed - let the natural 3σ bounds determine the size
    // The radii are already computed from the eigenvalues with 3σ scaling
    
    // Store quad parameters (all in NDC space)
    quad_params[quad_offset + 0] = ndc_x;
    quad_params[quad_offset + 1] = ndc_y;
    quad_params[quad_offset + 2] = radius_x_ndc;
    quad_params[quad_offset + 3] = radius_y_ndc;
    quad_params[quad_offset + 4] = ndc_z;  // Store NDC z for depth
    
    // All quads are visible (already set above)
}

// Compact visible quads using prefix sum - maintains sorted order
extern "C" __global__
void compact_visible_quads(const float* quad_vertices_in,       // Input quad vertices (all quads)
                          const float* quad_uvs_in,            // Input UV coordinates (all quads)  
                          const float* quad_data_in,           // Input quad data (all quads)
                          const int* visibility_mask,         // Visibility mask
                          const int* prefix_sum,               // Prefix sum of visibility mask
                          float* quad_vertices_out,            // Output: compacted quad vertices
                          float* quad_uvs_out,                 // Output: compacted UV coordinates
                          float* quad_data_out,                // Output: compacted quad data
                          int num_points) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= num_points) return;
    
    // Skip if not visible
    if (visibility_mask[idx] == 0) return;
    
    // Get output index from prefix sum (maintains sorted order)
    int output_idx = prefix_sum[idx];
    
    // Copy quad vertices (4 vertices per quad)
    for (int v = 0; v < 4; v++) {
        int input_vertex_idx = idx * 4 + v;
        int output_vertex_idx = output_idx * 4 + v;
        
        // Copy vertex data (8 floats per vertex: x,y,z,r,g,b,center_x,center_y)
        for (int f = 0; f < 8; f++) {
            quad_vertices_out[output_vertex_idx * 8 + f] = quad_vertices_in[input_vertex_idx * 8 + f];
        }
        
        // Copy UV data (2 floats per vertex: u,v)
        for (int f = 0; f < 2; f++) {
            quad_uvs_out[output_vertex_idx * 2 + f] = quad_uvs_in[input_vertex_idx * 2 + f];
        }
    }
    
    // Copy quad data (6 floats per quad: opacity, inv_cov components, radii)
    for (int f = 0; f < 6; f++) {
        quad_data_out[output_idx * 6 + f] = quad_data_in[idx * 6 + f];
    }
}

// Generate quad vertices for visible Gaussians
extern "C" __global__
void generate_quad_vertices(const float* quad_params,           // Quad parameters (center_x, center_y, radius_x, radius_y, ndc_z)
                           const float* cov2d_data,             // 2D covariance matrices (3 components each)
                           const int* visibility_mask,         // Visibility mask
                           const float* colors,                // Colors (3 components each)
                           const float* opacities,             // Opacity values
                           float* quad_vertices,               // Output: Quad vertices (8 floats per vertex: x,y,z,r,g,b,center_x,center_y)
                           float* quad_uvs,                    // Output: UV coordinates (2 floats per vertex)
                           float* quad_data,                   // Output: Per-quad data (opacity + 2D covariance inverse)
                           int* visible_count,                 // Output: Number of visible quads
                           int num_points) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= num_points) return;
    
    // Load quad parameters - no visibility check for performance
    int param_offset = idx * 5;
    float center_x = quad_params[param_offset + 0];
    float center_y = quad_params[param_offset + 1];
    float radius_x = quad_params[param_offset + 2];
    float radius_y = quad_params[param_offset + 3];
    float ndc_z = quad_params[param_offset + 4];
    
    // Use the original sorted index to maintain depth order
    // Don't use atomic counter as it breaks sorting
    int quad_idx = idx;
    
    
    // Load 2D covariance matrix
    int cov_offset = idx * 3;
    float cov_00 = cov2d_data[cov_offset + 0];
    float cov_01 = cov2d_data[cov_offset + 1];
    float cov_11 = cov2d_data[cov_offset + 2];
    
    // Compute inverse of 2D covariance matrix for fragment shader
    float det = cov_00 * cov_11 - cov_01 * cov_01;
    
    
    // Skip if covariance matrix is degenerate (made more permissive)
    if (det <= 1e-12f || cov_00 <= 1e-12f || cov_11 <= 1e-12f) {
        return;
    }
    
    float inv_det = 1.0f / det;
    float inv_cov_00 = cov_11 * inv_det;
    float inv_cov_01 = -cov_01 * inv_det;
    float inv_cov_11 = cov_00 * inv_det;
    
    
    // Load color and opacity
    int color_offset = idx * 3;
    float r = colors[color_offset + 0];
    float g = colors[color_offset + 1];
    float b = colors[color_offset + 2];
    float opacity = opacities[idx];
    
    // Compute eigenvectors for oriented quad
    // For symmetric matrix [[a,b],[b,c]], eigenvector of larger eigenvalue:
    float trace = cov_00 + cov_11;
    float discriminant = trace * trace - 4.0f * det;
    float sqrt_disc = sqrtf(fmaxf(discriminant, 0.0f));
    float lambda1 = 0.5f * (trace + sqrt_disc);
    
    // Eigenvector corresponding to lambda1
    float evec_x, evec_y;
    if (fabsf(cov_01) > 1e-6f) {
        evec_x = lambda1 - cov_11;
        evec_y = cov_01;
        float norm = sqrtf(evec_x * evec_x + evec_y * evec_y);
        evec_x /= norm;
        evec_y /= norm;
    } else {
        // Diagonal matrix case
        evec_x = 1.0f;
        evec_y = 0.0f;
    }
    
    // Generate 4 vertices for the quad
    // Vertex layout: bottom-left, bottom-right, top-left, top-right
    float offsets_x[4] = {-1.0f, 1.0f, -1.0f, 1.0f};
    float offsets_y[4] = {-1.0f, -1.0f, 1.0f, 1.0f};
    float uvs[8] = {0.0f, 0.0f, 1.0f, 0.0f, 0.0f, 1.0f, 1.0f, 1.0f};
    
    for (int i = 0; i < 4; i++) {
        int vertex_idx = quad_idx * 4 + i;
        int vertex_offset = vertex_idx * 8;  // 8 floats per vertex (x,y,z,r,g,b,center_x,center_y)
        int uv_offset = vertex_idx * 2;      // 2 floats per vertex (u,v)
        
        // Rotate and scale offset by covariance eigenvectors
        float local_x = offsets_x[i] * radius_x;
        float local_y = offsets_y[i] * radius_y;
        
        float world_x = evec_x * local_x - evec_y * local_y;
        float world_y = evec_y * local_x + evec_x * local_y;
        
        // Store vertex position (in NDC space with proper depth)
        quad_vertices[vertex_offset + 0] = center_x + world_x;
        quad_vertices[vertex_offset + 1] = center_y + world_y;
        quad_vertices[vertex_offset + 2] = ndc_z;  // Use the NDC z-coordinate for proper depth
        
        // Store vertex color normally
        quad_vertices[vertex_offset + 3] = r;
        quad_vertices[vertex_offset + 4] = g;
        quad_vertices[vertex_offset + 5] = b;
        
        // Store Gaussian center position in NDC space (new!)
        quad_vertices[vertex_offset + 6] = center_x;
        quad_vertices[vertex_offset + 7] = center_y;
        
        // Store UV coordinates
        quad_uvs[uv_offset + 0] = uvs[i * 2 + 0];
        quad_uvs[uv_offset + 1] = uvs[i * 2 + 1];
    }
    
    // Store per-quad data for fragment shader
    // Extend to 6 components: opacity, inv_cov (3), radii (2)
    int quad_data_offset = quad_idx * 6;
    quad_data[quad_data_offset + 0] = opacity;
    quad_data[quad_data_offset + 1] = inv_cov_00;
    quad_data[quad_data_offset + 2] = inv_cov_01;
    quad_data[quad_data_offset + 3] = inv_cov_11;
    quad_data[quad_data_offset + 4] = radius_x;  // NDC radius in X direction
    quad_data[quad_data_offset + 5] = radius_y;  // NDC radius in Y direction
}

// Generate indices for rendering quads as triangles
extern "C" __global__
void generate_quad_indices(unsigned int* indices,    // Output: Triangle indices
                          int num_quads) {           // Number of quads to generate indices for
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= num_quads) return;
    
    // Each quad needs 6 indices (2 triangles)
    // Quad vertices are ordered: bottom-left(0), bottom-right(1), top-left(2), top-right(3)
    int base_vertex = idx * 4;
    int base_index = idx * 6;
    
    // First triangle: bottom-left, bottom-right, top-left (0, 1, 2)
    indices[base_index + 0] = base_vertex + 0;
    indices[base_index + 1] = base_vertex + 1;
    indices[base_index + 2] = base_vertex + 2;
    
    // Second triangle: bottom-right, top-right, top-left (1, 3, 2)
    indices[base_index + 3] = base_vertex + 1;
    indices[base_index + 4] = base_vertex + 3;
    indices[base_index + 5] = base_vertex + 2;
}

// Generate instance data for instanced rendering
// Each instance represents one Gaussian with all its properties
extern "C" __global__
void generate_instance_data(const float* quad_params,     // Quad parameters (center_x, center_y, radius_x, radius_y, ndc_z)
                           const float* cov2d_data,       // 2D covariance matrices (3 components each)
                           const int* visibility_mask,    // Visibility mask
                           const float* colors,           // Colors (3 components each)
                           const float* opacities,        // Opacity values
                           float* instance_data,          // Output: Instance data (10 floats per instance)
                           int num_points) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= num_points) return;
    
    // Load quad parameters - no visibility check
    int param_offset = idx * 5;
    float center_x = quad_params[param_offset + 0];
    float center_y = quad_params[param_offset + 1];
    float radius_x = quad_params[param_offset + 2];
    float radius_y = quad_params[param_offset + 3];
    float ndc_z = quad_params[param_offset + 4];
    
    // Load 2D covariance matrix
    int cov_offset = idx * 3;
    float cov_00 = cov2d_data[cov_offset + 0];
    float cov_01 = cov2d_data[cov_offset + 1];
    float cov_11 = cov2d_data[cov_offset + 2];
    
    // Compute inverse of 2D covariance matrix
    float det = cov_00 * cov_11 - cov_01 * cov_01;
    if (det <= 1e-12f) det = 1e-12f;  // Ensure non-zero determinant
    
    float inv_det = 1.0f / det;
    float inv_cov_00 = cov_11 * inv_det;
    float inv_cov_01 = -cov_01 * inv_det;
    float inv_cov_11 = cov_00 * inv_det;
    
    // Load color and opacity
    int color_offset = idx * 3;
    float r = colors[color_offset + 0];
    float g = colors[color_offset + 1];
    float b = colors[color_offset + 2];
    float opacity = opacities[idx];
    
    // Pack instance data (10 floats per instance)
    // Layout: center_x, center_y, ndc_z, r, g, b, opacity, inv_cov_00, inv_cov_01, inv_cov_11
    int instance_offset = idx * 10;
    instance_data[instance_offset + 0] = center_x;
    instance_data[instance_offset + 1] = center_y;
    instance_data[instance_offset + 2] = ndc_z;
    instance_data[instance_offset + 3] = r;
    instance_data[instance_offset + 4] = g;
    instance_data[instance_offset + 5] = b;
    instance_data[instance_offset + 6] = opacity;
    instance_data[instance_offset + 7] = inv_cov_00;
    instance_data[instance_offset + 8] = inv_cov_01;
    instance_data[instance_offset + 9] = inv_cov_11;
}